
#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ void myKernel(){
    printf("Hello, world from the device!\n");
}

__global__ void llama_gemv(){
  myKernel();
}

int main_cu()
{
    llama_gemv<<<4,4>>>();
    hipError_t hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(hipError_t));
        return 1;
        }
        else {
            printf("No CUDA error\n");
            }
     hipDeviceSynchronize();
     return 0;
}
